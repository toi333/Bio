#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <CudaAligner/CudaAligner.h>
#include <CudaAligner/Util.h>

#include <Base/Core.h>

#include <iostream>

struct LocalRowData
{
    int y;
    int row;
    int valBestInCol;
};

__global__ void kernel(int iBlock0, int iBlockRun, int blockRunProgress, int ctColPerThread, int ctRow, int ctCol, int defVal,
    int *row, int *col, int *valBestInRow, int *valBestInCol, char *x, char *y, EndPoint *bests, const Scoring sc)
{
    extern __shared__ LocalRowData _lrd[];

    const int iBlock = iBlock0 + blockIdx.x;

    const int iThread0 = iBlock * blockDim.x;

    LocalRowData *lrd = _lrd - ctColPerThread * iThread0;

    const int iThreadInGrid = iThread0 + threadIdx.x;
    const int iCol0T = ctColPerThread * iThreadInGrid;
    const int iCol1T = ctColPerThread * (iThreadInGrid + 1);

    EndPoint best;
    best.val = INT_MIN;
    best.p = Vec2i{ -1, -1 };

    const int blockOffset = blockRunProgress * (iBlockRun - iBlock) - iBlock - iBlock * 2 * ((int)blockDim.x - 1);

    const int threadCurRowOffset = -2 * (int)threadIdx.x;

    const int iRow0T = blockOffset + threadCurRowOffset;
    const int iRow1T = blockOffset + blockRunProgress + threadCurRowOffset;

    const int iRow1S = min(iRow1T, ctRow);
    
    __syncthreads();

    if (iRow0T < ctRow) {
        for (int iCol = iCol0T; iCol < iCol1T; ++iCol) {
            lrd[iCol].row = row[iCol];
            lrd[iCol].valBestInCol = valBestInCol[iCol];
            lrd[iCol].y = y[iCol];
        }
    }

    __syncthreads();

    for (int iRow = iRow0T; iRow != iRow1T; ++iRow) {
        if (0 <= iRow && iRow < ctRow) {
            int prevColRow = col[iRow - 1];
            // TODO: fix this
            int prevCol = iRow == ctRow - 1 && iThreadInGrid != 0 ? row[iCol0T - 1] : col[iRow];
            int valBestInRowLocal = valBestInRow[iRow];
            char xLocal = x[iRow];
            for (int iCol = iCol0T; iCol < iCol1T; ++iCol) {
                int r = defVal;

                const int prevRow = lrd[iCol].row;
                valBestInRowLocal = max(valBestInRowLocal - sc.k, prevCol - sc.b);
                lrd[iCol].valBestInCol = max(lrd[iCol].valBestInCol - sc.k, prevRow - sc.b);

                r = max(r, valBestInRowLocal);
                r = max(r, lrd[iCol].valBestInCol);

                //r = max(r, prevColRow + sc.match(ca->dX[iRow + rev], ca->dY[iCol + rev]));
                r = max(r, prevColRow + (xLocal == lrd[iCol].y ? sc.mp : sc.mn));

                prevColRow = prevRow;
                lrd[iCol].row = r;
                prevCol = r;

                if (r > best.val && iCol < ctCol) {
                    best.val = r;
                    best.p = Vec2i{ iRow, iCol };
                }
            }
            col[iRow - 1] = prevColRow;
            valBestInRow[iRow] = valBestInRowLocal;
            if (iRow == iRow1S - 1) {
                for (int iCol = iCol0T; iCol < iCol1T; ++iCol) {
                    row[iCol] = lrd[iCol].row;
                    valBestInCol[iCol] = lrd[iCol].valBestInCol;
                }
            }
        }
        __syncthreads();
    }

    // TODO: beter init
    if (iBlockRun == 0 || best.val > bests[iThreadInGrid].val) {
        bests[iThreadInGrid] = best;
    }
} 

int callKernel(int ctRow, int ctCol, char *x, char *y, int defVal, CudaAligner *ca, const Scoring &sc)
{
    const int ctColPerThread = 4;
    const int ctThreadPerFullBlock = 1024;
    const int blockRunProgress = 1024;

    const int ctTotalThreads = (ctCol + ctColPerThread - 1) / ctColPerThread;

    const int ctBlocks = (ctTotalThreads + ctThreadPerFullBlock - 1) / ctThreadPerFullBlock;

    const int ctThreadsPerBlock = (ctTotalThreads + ctBlocks - 1) / ctBlocks;

    const int ctColR = ctBlocks * ctThreadsPerBlock * ctColPerThread;

    const int iLastThreadInBlock = ctThreadsPerBlock - 1;
    const int iLastBlock = ctBlocks - 1;

    const int lastThreadOffset = 2 * iLastThreadInBlock;

    const int ctBlockRuns = (ctRow + blockRunProgress * iLastBlock + iLastBlock + iLastBlock * 2 * iLastThreadInBlock + 2 * iLastThreadInBlock + blockRunProgress - 1) / blockRunProgress;

    for (int iBlockRun = 0; iBlockRun < ctBlockRuns; ++iBlockRun) {
        //const int iBlock0 = min(max(0, (blockRunProgress * iBlockRun - ctRow) / (blockRunProgress + 1)), ctBlocks - 1);
        const int iBlock0 = 0;
        //const int iBlock1 = min(max(0, (blockRunProgress * (iBlockRun + 2)) / (blockRunProgress + 1)), ctBlocks - 1);
        const int iBlock1 = ctBlocks - 1;

        const int ctBlocksTrim = iBlock1 - iBlock0 + 1;

        kernel<<<ctBlocksTrim, ctThreadsPerBlock, ctThreadPerFullBlock * ctColPerThread  * sizeof(LocalRowData)>>>(
            iBlock0, iBlockRun, blockRunProgress, ctColPerThread, ctRow, ctCol, defVal,
            ca->dRow, ca->dCol, ca->dValBestInRow, ca->dValBestInCol, x, y, ca->dBest, sc);

        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipDeviceSynchronize());
    }

    return ctTotalThreads;
}
