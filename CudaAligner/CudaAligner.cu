#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <CudaAligner/CudaAligner.h>
#include <CudaAligner/Util.h>

#include <Base/Core.h>

struct LocalRowData
{
    int y;
    int row;
    int valBestInCol;
};

__global__ void kernel(int iBlockRun, int blockRunProgress, int ctColPerThread, int ctRow, int ctCol, int ctColReal, int defVal,
    int *row, int *col, int *valBestInRow, int *valBestInCol, char *x, char *y, EndPoint *bests, const Scoring sc)
{
    extern __shared__ LocalRowData _lrd[];

    const int iThread0 = blockIdx.x * blockDim.x;

    LocalRowData *lrd = _lrd - ctColPerThread * iThread0;

    const int iThreadInGrid = iThread0 + threadIdx.x;
    const int iCol0T = ctColPerThread * iThreadInGrid;
    const int iCol1T = ctColPerThread * (iThreadInGrid + 1);

    EndPoint best;
    best.val = INT_MIN;
    best.p = Vec2i{ -1, -1 };

    const int blockOffset = blockRunProgress * (iBlockRun - (int)blockIdx.x) - (int)blockIdx.x;

    const int iRow0B = blockOffset;
    const int iRow1B = iRow0B + blockRunProgress;

    const int threadRowProgress = 1;
    const int threadCurRowOffset = threadRowProgress * (-(int)threadIdx.x) - (int)threadIdx.x;
    const int lastThreadCurRowOffset = threadRowProgress * (-((int)blockDim.x - 1)) - ((int)blockDim.x - 1);

    const int iRow0T = iRow0B + threadCurRowOffset;
    const int iRow1T = iRow1B + threadCurRowOffset - lastThreadCurRowOffset;

    const int iRow0S = max(0, iRow0B);
    const int iRow1S = min(ctRow, iRow1B);
    
    __syncthreads();

    if (iRow0T < ctRow) {
        for (int iCol = iCol0T; iCol < iCol1T; ++iCol) {
            lrd[iCol].row = row[iCol];
            lrd[iCol].valBestInCol = valBestInCol[iCol];
            lrd[iCol].y = y[iCol];
        }
    }

    __syncthreads();

    for (int iRow = iRow0T; iRow != iRow1T; ++iRow) {
        if (iRow0S <= iRow && iRow < iRow1S) {
            int prevColRow = col[iRow - 1];
            // TODO: fix this
            const bool end = iRow == iRow1S - 1 || iRow == iRow1S - 1;
            int prevCol = end && iThreadInGrid != 0 ? row[iCol0T - 1] : col[iRow];
            int valBestInRowLocal = valBestInRow[iRow];
            char xLocal = x[iRow];
            for (int iCol = iCol0T; iCol < iCol1T; ++iCol) {
                int r = defVal;

                const int prevRow = lrd[iCol].row;
                valBestInRowLocal = max(valBestInRowLocal - sc.k, prevCol - sc.b);
                lrd[iCol].valBestInCol = max(lrd[iCol].valBestInCol - sc.k, prevRow - sc.b);

                r = max(r, valBestInRowLocal);
                r = max(r, lrd[iCol].valBestInCol);

                //r = max(r, prevColRow + sc.match(ca->dX[iRow + rev], ca->dY[iCol + rev]));
                r = max(r, prevColRow + (xLocal == lrd[iCol].y ? sc.mp : sc.mn));

                prevColRow = prevRow;
                lrd[iCol].row = r;
                prevCol = r;

                if (r > best.val && iCol < ctColReal) {
                    best.val = r;
                    best.p = Vec2i{ iRow, iCol };
                }
            }
            col[iRow - 1] = prevColRow;
            valBestInRow[iRow] = valBestInRowLocal;
            if (end) {
                for (int iCol = iCol0T; iCol < iCol1T; ++iCol) {
                    row[iCol] = lrd[iCol].row;
                    valBestInCol[iCol] = lrd[iCol].valBestInCol;
                }
            }
        }
        __syncthreads();
    }

    // TODO: beter init
    if (iBlockRun == 0 || best.val > bests[iThreadInGrid].val) {
        bests[iThreadInGrid] = best;
    }
} 

int callKernel(int ctRow, int ctCol, char *x, char *y, int defVal, CudaAligner *ca, const Scoring &sc)
{
    const int ctColPerThread = 4;
    const int ctThreadPerFullBlock = 1024;
    const int blockRunProgress = 1024;

    const int ctTotalThreads = (ctCol + ctColPerThread - 1) / ctColPerThread;

    const int ctBlocks = (ctTotalThreads + ctThreadPerFullBlock - 1) / ctThreadPerFullBlock;

    const int ctThreadsPerBlock = (ctTotalThreads + ctBlocks - 1) / ctBlocks;

    const int ctColR = ctBlocks * ctThreadsPerBlock * ctColPerThread - 1;

    const int iLastBlock = ctBlocks - 1;
    const int ctBlockRuns = ((ctRow + iLastBlock) + blockRunProgress - 1) / blockRunProgress + iLastBlock;

    for (int iBlockRun = 0; iBlockRun < ctBlockRuns; ++iBlockRun) {
        kernel<<<ctBlocks, ctThreadsPerBlock, ctThreadPerFullBlock * ctColPerThread  * sizeof(LocalRowData)>>>(
            iBlockRun, blockRunProgress, ctColPerThread, ctRow, ctColR, ctCol, defVal,
            ca->dRow, ca->dCol, ca->dValBestInRow, ca->dValBestInCol, x, y, ca->dBest, sc);

        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipDeviceSynchronize());
    }

    return ctTotalThreads;
}
